#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void print_hello_cuda() {
	int i = threadIdx.x;
	printf("[PRINT] ThreadId.x: %d \n", i);
}

int main() {
	print_hello_cuda << <1, 8 >> > ();
	return 0;
}